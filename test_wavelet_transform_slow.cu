#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define VERBOSE 0
#define NVCC
#include "printing.cuh"

#include "cuda_err_check.h"
#include "read_volume.h"
#include "write_volume.h"
#include "wavelet_slow.h"
#include "wavelet_slow.cuh"
#include "opt_32_6.cuh"
#include "compare.h"
#include "diff.h"
#include "norms.h"
#include "init_x.h"
#include "init_random.h"

const int FORWARD = 0;
const int INVERSE = 1;
const int CPU_COMPUTE = 0;
const int ERR_CHECK = 1;

const int RUN_32x32x32 = 1;
const int RUN_8x8x8 = 0;


int main(int argc, char **argv) {

        
        const char *filename = argv[1];
        const char *outfilename = argv[2];
        
        int nx, ny, nz, bx, by, bz;
        float *x, *x2;

        if (filename) {
                printf("reading: %s \n", filename);
                read_volume(filename, x, nx, ny, nz, bx, by, bz);
                read_volume(filename, x2, nx, ny, nz, bx, by, bz);
        } else {

                nx = 32;
                ny = 32; 
                nz = 32;
                bx = 1;
                by = 1;
                bz = 1;

                init_random(x, nx, ny, nz, bx, by, bz);


        }
        printf("block dimension: %d %d %d \n", nx, ny, nz);
        printf("number of blocks: %d %d %d \n", bx, by, bz);
        size_t num_bytes = sizeof(float) * nx * ny * nz * bx * by * bz;
        float *work = (float*)malloc(num_bytes);
        float *x_gpu = (float*)malloc(num_bytes);
        float *err = (float*)malloc(num_bytes);

        float *d_x;
        hipMalloc((void**)&d_x, num_bytes);
        hipMemcpy(d_x, x2, num_bytes, hipMemcpyHostToDevice);

        int x0 = 0;
        int y0 = 0;
        int z0 = 0;

        int b = bx * by * bz;
        int n = nx * ny * nz;


        if (CPU_COMPUTE) {

        printf("Computing CPU forward transform (single block) ... \n");
        Wavelet_Transform_Slow_Forward(x, work, 32, 32, 32, x0, y0, z0, 32, 32, 32);

        printf("Computing CPU inverse transform (single block) ... \n");
        Wavelet_Transform_Slow_Inverse(x, work, 32, 32, 32, x0, y0, z0, 32, 32, 32);

        const char *errtype[] = {"abs.", "rel."};
        for (int a = 0; a < 2; ++a) {
        double l2err = l2norm(x, x2, b * n, a);
        double l1err = l1norm(x, x2, b * n, a);
        double linferr = linfnorm(x, x2, b * n, a);
        printf("%s l2 error = %g l1 error = %g linf error = %g \n", errtype[a], l2err, l1err, linferr);
        }
        }

        if (RUN_32x32x32) {

        hipEvent_t start, stop;
        float elapsed = 0;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        printf("[32, 32, 32] Computing GPU forward transform... \n");
        hipEventRecord(start);
        opt6wl79_32x32x32_h<FORWARD>(d_x, bx, by, bz);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);

        hipDeviceSynchronize();
        printf("Throughput: %g Mcells/s \n", b * n / elapsed / 1e3); 

        printf("[32, 32, 32] Computing GPU inverse transform... \n");
        hipEventRecord(start);
        wl79_32x32x32_h<INVERSE>(d_x, bx, by, bz);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        printf("Throughput: %g Mcells/s \n", b * n / elapsed / 1e3); 

        if (ERR_CHECK) {
                printf("Running error checking... \n");
                hipMemcpy(x_gpu, d_x, num_bytes, hipMemcpyDeviceToHost);

                //print_array(x_gpu, 32, 32, 32, 0, 0, 0, 4, 4, 4);
                //print_array(x, 32, 32, 32, 0, 0, 0, 4, 4, 4);
                //assert(compare(x, x_gpu, 8, 8, 8, 1));

                const char *errtype[] = {"abs.", "rel."};
                for (int a = 0; a < 2; ++a) {
                double l2err = l2norm(x2, x_gpu, b * n, a);
                double l1err = l1norm(x2, x_gpu, b * n, a);
                double linferr = linfnorm(x2, x_gpu, b * n, a);
                printf("%s l2 error = %g l1 error = %g linf error = %g \n", errtype[a], l2err, l1err, linferr);
                }
        }
        }


        if (RUN_8x8x8) {

        cudaErrCheck(hipMemcpy(d_x, x2, num_bytes, hipMemcpyHostToDevice));

        hipEvent_t start, stop;
        float elapsed = 0;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        printf("[8, 8, 8] Computing GPU forward transform... \n");
        hipEventRecord(start);
        wl79_8x8x8_h<FORWARD>(d_x, bx, by, bz);
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipEventElapsedTime(&elapsed, start, stop);
        hipDeviceSynchronize();
        printf("Throughput: %g Mcells/s \n", b * n / elapsed / 1e3); 


        printf("[8, 8, 8] Computing GPU inverse transform... \n");
        elapsed = 0;
        hipEventRecord(start);
        wl79_8x8x8_h<INVERSE>(d_x, bx, by, bz);
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipEventElapsedTime(&elapsed, start, stop);
        hipDeviceSynchronize();
        printf("Throughput: %g Mcells/s \n", b * n / elapsed / 1e3); 
        hipDeviceSynchronize();

        if (ERR_CHECK) {
                printf("Running error checking... \n");
                hipMemcpy(x_gpu, d_x, num_bytes, hipMemcpyDeviceToHost);
                assert(compare(x, x_gpu, 8, 8, 8, 1, 1e-3f));

                const char *errtype[] = {"abs.", "rel."};
                for (int a = 0; a < 2; ++a) {
                double l2err = l2norm(x2, x_gpu, b * n, a);
                double l1err = l1norm(x2, x_gpu, b * n, a);
                double linferr = linfnorm(x2, x_gpu, b * n, a);
                printf("%s l2 error = %g l1 error = %g linf error = %g \n", errtype[a], l2err, l1err, linferr);
                }
        }

        if (VERBOSE) {
                diff(err, x, x_gpu, 512);
                printf("x = \n");
                print_array(x, 8, 8, 8);
                printf("x_gpu = \n");
                print_array(x_gpu, 8, 8, 8);
                printf("err = \n");
                print_array(err, 8, 8, 8);
        }
        }
        
        printf("Test(s) passed!\n");


        if (outfilename) {
                printf("writing: %s \n", outfilename);
                write_volume(outfilename, x, nx, ny, nz, bx, by, bz);
        }

        
}

